
#include <hip/hip_runtime.h>
#include <ctime> 
#include <cstdlib> 
#include <iostream>
#include <string>
#include <cmath>
#include <vector>

class Pt
{
public:
	float x = 0;
	float y = 0;
	int group = 1;
};
__global__ void setFalse(bool*& Changed, int dsize);
__device__ float dist(const Pt& p1, const Pt& p2);
__global__ void Group_find(Pt*& data, int dsize, Pt* dev_ctrs, bool*& moved);
__global__ void Moved_find(bool* moved, int dsize, bool* dev_isMoved);
__global__ void Group_update(Pt*& data, int dsize, float* sums, int* cnts);

int main()
{
	unsigned seed = time(0);
	srand(seed);

	int dsize=10;
	printf("enter data size  ");
	scanf("%d", &dsize);
	const int gsize = dsize/2;
	const int m1 = 0, n1 = gsize;
	const int m2 = n1+1, n2 = dsize;

	Pt expected1, expected2;
	float xsum = 0, ysum = 0;

	Pt* data;
	hipMallocManaged( &data, dsize * sizeof(Pt) );
	bool* moved;
	hipMallocManaged( &moved, dsize * sizeof(bool) );

	int blockSize = 1024;
	int blockNum = (dsize + blockSize - 1) / blockSize;

	Pt* dataTemp  = new Pt[dsize];
	for(int i = 0; i < gsize; ++i)
	{
		Pt p;
		p.x = m1 + rand() % (n1 - m1);
		xsum += p.x;
		p.y = m1 + rand() % (n1 - m1);
		ysum += p.y;
		dataTemp[i]=p;
	}
	expected1.x = xsum/gsize;
	expected1.y = ysum/gsize;


	xsum = 0, ysum = 0;
	for(int i = 0; i < gsize; ++i)
	{
		Pt p;
		p.x = m2 + rand() % (n2 - m2);
		xsum += p.x;
		p.y = m2 + rand() % (n2 - m2);
		ysum += p.y;
		dataTemp[i + gsize]=p;
	}
	expected2.x = xsum/gsize;
	expected2.y = ysum/gsize;
	

	hipMemcpy(data,dataTemp, dsize * sizeof( Pt ), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	Pt* ctrs = new Pt[2]; 
	ctrs[0].x = m1 + rand() % (n2-m1);
	ctrs[0].y = m1 + rand() % (n2-m1);
	ctrs[1].x = m1 + rand() % (n2-m1);
	ctrs[1].y = m1 + rand() % (n2-m1);

	Pt* dev_ctrs;
	hipMallocManaged(&dev_ctrs, 2 * sizeof(Pt));
	hipMemcpy(dev_ctrs, ctrs, 2 * sizeof( Pt ), hipMemcpyHostToDevice);

	float* sums = new float[4];
	for(int s = 0; s < 4; ++s) sums[s] = 0;
	float* dev_sums;
	hipMallocManaged(&dev_sums, 4 * sizeof(float));

	int* cnts = new int[2];
	cnts[0] = 1; cnts[1] = 1;
	int* dev_cnts;
	hipMallocManaged(&dev_cnts, 2 * sizeof(int));


	bool* isMoved = new bool[1]; 
	isMoved[0] = true;

	bool* dev_isMoved;
	hipMallocManaged(&dev_isMoved, sizeof(bool));

	while( isMoved[0] )
	{
		printf("Center1 = ( %.2f, %.2f )\n", ctrs[0].x ,ctrs[0].y);
		printf("Center2 = ( %.2f, %.2f )\n", ctrs[1].x, ctrs[1].y);
		isMoved[0] = false;

		clock_t st = clock();
		setFalse<<<blockNum, blockSize>>>(moved, dsize);
		hipDeviceSynchronize();
		Group_find<<<blockNum, blockSize>>>(data, dsize, dev_ctrs, moved);
		hipDeviceSynchronize();
		hipMemcpy(dev_isMoved, isMoved, sizeof( bool ), hipMemcpyHostToDevice);

		Moved_find<<<1, 1>>>(moved, dsize, dev_isMoved);
		hipDeviceSynchronize();
		hipMemcpy(isMoved, dev_isMoved, sizeof( bool ), hipMemcpyDeviceToHost);

		clock_t st2 = clock();
		clock_t st3 = 0;
		clock_t st4 = 0;
		if( isMoved[0] )
		{
			st3 = clock();
			hipMemcpy(dev_sums, sums, 4 * sizeof( float ), hipMemcpyHostToDevice);
			hipMemcpy(dev_cnts, cnts, 2 * sizeof( int ), hipMemcpyHostToDevice);

			Group_update<<<blockNum, blockSize>>>(data, dsize, dev_sums, dev_cnts);
			hipDeviceSynchronize();

			hipMemcpy(sums, dev_sums, 4 * sizeof( float ), hipMemcpyDeviceToHost);
			hipMemcpy(cnts, dev_cnts, 2 * sizeof( int ), hipMemcpyDeviceToHost);
			st4 = clock();
			ctrs[0].x = sums[0] / cnts[0];
			ctrs[0].y = sums[1] / cnts[0];
			ctrs[1].x = sums[2] / cnts[1];
			ctrs[1].y = sums[3] / cnts[1];
		}
		clock_t st5 = clock();
		hipMemcpy(ctrs,dev_ctrs, 2 * sizeof( Pt ), hipMemcpyDeviceToHost);
	
	printf("\n Elapsed Time : %u ms \n", clock() - st5 + (st4 - st3) + (st2 - st));

	}
	
	printf("---Result---:\n");
	printf("Expected1 = ( %.2f, %.2f )\n",expected1.x, expected1.y);
	printf("Expected2 = ( %.2f, %.2f )\n", expected2.x, expected2.y);

	printf("random initial Center1 = ( %.2f, %.2f )" ,ctrs[0].x ,ctrs[0].y);
	printf("random initial Center2 = ( %.2f, %.2f )", ctrs[1].x, ctrs[1].y);

	hipFree(&data);
	hipFree(&moved);
	delete [] dataTemp;
	delete [] isMoved;
	hipFree( &dev_isMoved);

	delete [] sums;
	hipFree( &dev_sums);
	delete [] cnts;
	hipFree( &dev_cnts);

}

__device__ float dist(const Pt& p1, const Pt& p2)
{
	float s = sqrt(pow((p1.x - p2.x), 2) + pow((p1.y - p2.y), 2));
	return s;
}

__global__ void setFalse(bool*& Changed, int dsize)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < dsize)
	{
		Changed[index] = false;
	}
}

__global__ void Group_find(Pt*& data, int dsize, Pt* dev_ctrs, bool*& moved)
{
	int p = blockIdx.x * blockDim.x + threadIdx.x;

	if (p < dsize)
	{
		float d1 = dist(dev_ctrs[0], data[p]);
		float d2 = dist(dev_ctrs[1], data[p]);
		int oldGroup = data[p].group;

		if (d1 < d2)
			data[p].group = 1;
		else
			data[p].group = 2;

		if (data[p].group != oldGroup)
		{
			moved[p] = true;
		}
	}
}

__global__ void Moved_find(bool* moved, int dsize, bool* dev_isMoved)
{
	int index = 0;
	while (index < dsize && !dev_isMoved[0])
	{
		if (moved[index] == true) {
			dev_isMoved[0] = true;
		}
		index++;
	}
}

__global__ void Group_update(Pt*& data, int dsize, float* sums, int* cnts)
{

	int p = blockIdx.x * blockDim.x + threadIdx.x;

	if (p < dsize)
	{
		if (data[p].group == 1)
		{
			sums[0] += data[p].x; sums[1] += data[p].y;
			cnts[0]++;
		}
		else
		{
			sums[2] += data[p].x; sums[3] += data[p].y;
			cnts[1]++;
		}
	}
}
